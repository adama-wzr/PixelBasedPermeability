#include "hip/hip_runtime.h"
#include "Perm2D.cuh"

int main(void){

	// More efficient printing with parallel computing/Linux
	fflush(stdout);

	// Parse user entered options

	options opts;	// struct to hold options
	simulationInfo simInfo;		// Struct to hold constants, variables, and results intrinsic to the simulation

	char inputFilename[100];

	sprintf(inputFilename, "input.txt");

	readInputFile(inputFilename, &opts);

	// Read 2D Input Image

	int width, height, channel;
	unsigned char* targetImage;

	readImage(&targetImage, &width, &height, &channel, opts.inputFilename);

	if (channel != 1){
		printf("Error: please enter a grascale image with 1 channel.\n Current number of channels = %d\n", channel);
		return 1;
	}

	simInfo.porosity = calcPorosity(targetImage, width, height);

	if(opts.verbose == 1){
		std::cout << "Image Parameters:" << std::endl;
		std::cout <<  "\n--------------------------------------" << std::endl;
		std::cout << "Width (pixels) = " << width << " Height (pixels) = " << height << " Channel = " << channel << std::endl;
		std::cout << "Porosity = " << simInfo.porosity << std::endl;
	}

	// Define mesh related parameters

	simInfo.numCellsX = width*opts.MeshAmp;			// Simulation Grid width in number of cells
	simInfo.numCellsY = height*opts.MeshAmp;		// Simulation Grid height in number of cells
	simInfo.nElements = simInfo.numCellsY*simInfo.numCellsX;	// Number of elements (total)
	simInfo.dx = opts.DomainWidth/simInfo.numCellsX;			// dx
	simInfo.dy = opts.DomainWidth/simInfo.numCellsY;			// dy

	unsigned int *Grid = (unsigned int*)malloc(sizeof(int)*simInfo.numCellsX*simInfo.numCellsY);		// Array that will hold binary domain (solid vs fluid)

	// Mesh Amplify and decode image into binary matrix

	for(int i = 0; i<simInfo.numCellsY; i++){
		for (int j = 0; j<simInfo.numCellsX; j++){
			int targetIndex_Row = i/opts.MeshAmp;
			int targetIndex_Col = j/opts.MeshAmp;
			if(targetImage[targetIndex_Row*width + targetIndex_Col] < 150){
				Grid[i*simInfo.numCellsX + j] = 0; 			// black => fluid => 0 => void
			} else{
				Grid[i*simInfo.numCellsX + j] = 1;			// white => solid => 1 => material
			}
		}
	}

	// Flood Fill to eliminate non-participating media

	FloodFill(Grid, &simInfo);

	// Define arrays essential for the solution

	float *Pressure = (float *)malloc(sizeof(float)*simInfo.nElements);					// store pressure
	float *U = (float *)malloc(sizeof(float)*(simInfo.numCellsX+1)*simInfo.numCellsY);	// store U velocity
	float *V = (float *)malloc(sizeof(float)*(simInfo.numCellsY+1)*simInfo.numCellsX);	// store V velocity

	float *uExp = (float *)malloc(sizeof(float)*(simInfo.numCellsX+1)*simInfo.numCellsY);	// store explicit U velocity
	float *vExp = (float *)malloc(sizeof(float)*(simInfo.numCellsY+1)*simInfo.numCellsX);	// store explicit V velocity

	float *uCoeff = (float *)malloc(sizeof(float)*(simInfo.numCellsX+1)*simInfo.numCellsY);	// store U velocity coefficients
	float *vCoeff = (float *)malloc(sizeof(float)*(simInfo.numCellsY+1)*simInfo.numCellsX);	// store V velocity coefficients

	// Initialize arrays

	for(int row = 0; row<simInfo.numCellsY; row++){
		for(int col = 0; col< simInfo.numCellsX+1; col++){
			int index = row*(simInfo.numCellsX + 1) + col;
			if(col < simInfo.numCellsX){
				// Pressure[row*(simInfo.numCellsX) + col] = (opts.PL + opts.PR)/2;
				Pressure[row*(simInfo.numCellsX) + col] = opts.PR;
			}
			U[index] = 0.01;
			uExp[index] = 0.01;
			uCoeff[index] = 0;
			V[index] = 0;
			vCoeff[index] = 0;
			vExp[index] = 0;
		}	
	}

	// Now we use the SUV-CUT algorithm to solve velocity-pressure coupled

	float RMS = 1.0;
	long int iter = 0;

	float PermTHR = 0.001;

	float PermOld = 1;

	float PermChange = 1;

	FILE *OUT;

	OUT = fopen("ConvData.csv", "w");

	fprintf(OUT, "iter,K,R,alpha,mesh,Qavg\n");

	while(iter < opts.MaxIterGlobal && RMS > opts.ConvergenceRMS && PermChange > PermTHR){

		/*
			SUV-CUT procedure:
			- Solve for explicit component of u and v velocities
			- Use explicit u and v to solve for pressure implicitly
			- Use pressure solutions to correct u and v explicitly

			- (optional) solve equations of state to update physical properties

			Repeat until converged.

		*/

		if(iter == 0){
			printf("Global Iter: %ld\n\n", iter+1);
		}else if(iter % 20 == 0){
			printf("Global Iter: %ld\n", iter+1);
			printf("Permeability: %f\n", simInfo.Perm);
			printf("Continuity RMS: %1.9f\n", RMS);
			printf("Perm Change: %1.6f\n\n", PermChange);
		}

		explicitMomentum(Grid, uExp, vExp, U, V, uCoeff, vCoeff, &opts, &simInfo);

		implicitPressure(Grid, uExp, vExp, uCoeff, vCoeff, Pressure, &opts, &simInfo);

		momentumCorrection(Grid, uExp, vExp, U, V, uCoeff, vCoeff, Pressure, &opts, &simInfo);

		RMS = ResidualContinuity(U, V, &opts, &simInfo);

		PermCalc(U, &opts, &simInfo);

		fprintf(OUT, "%ld,%1.9f,%1.9f,%f,%d,%1.9f\n",iter,simInfo.Perm, RMS, opts.alphaRelax, opts.MeshAmp, simInfo.Flowrate);

		if(iter % 100 == 0){
			PermChange = fabs((simInfo.Perm - PermOld)/(simInfo.Perm));
			PermOld = simInfo.Perm;
		}

		iter++;
	}

	fclose(OUT);

	// ResMap(U, V, &opts, &simInfo);

	if(opts.printMaps == 1){
		printPUVmaps(Pressure, U, V, &opts, &simInfo);
	}
	
	// Housekeeping

	free(Pressure);
	free(U);
	free(V);
	free(uCoeff);
	free(vCoeff);
	free(uExp);
	free(vExp);
	free(Grid);

	return 0;
}