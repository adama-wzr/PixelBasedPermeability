#include "hip/hip_runtime.h"
#include "Perm2D.cuh"

int main(void){

	// More efficient printing with parallel computing/Linux
	fflush(stdout);

	// Parse user entered options

	bool convFlag = true;

	options opts;	// struct to hold options

	char inputFilename[100];

	sprintf(inputFilename, "input.txt");

	readInputFile(inputFilename, &opts);

	// Set OpenMP CPU environment

	int numThreads = opts.nCores;
	int nGPUs;

	omp_set_num_threads(numThreads);
	hipGetDeviceCount(&nGPUs);

	printf("Do I even get here?CPU = %d, GPU = %d\n", numThreads, nGPUs);

	// Let's pre allocate all arrays here globally and then distribute them

	int nRows = opts.MeshAmp*128;
	int nCols = opts.MeshAmp*128;
	int nElements = nRows*nCols;

	unsigned int *Global_Grid = (unsigned int*)malloc(sizeof(int)*nElements*nGPUs);

	float *Global_Pressure = (float *)malloc(sizeof(float)*nElements*nGPUs);					// store pressure
	float *Global_U = (float *)malloc(sizeof(float)*(nCols+1)*nRows*nGPUs);	// store U velocity
	float *Global_V = (float *)malloc(sizeof(float)*(nCols+1)*nRows*nGPUs);	// store V velocity

	float *Global_uExp = (float *)malloc(sizeof(float)*(nCols+1)*nRows*nGPUs);	// store explicit U velocity
	float *Global_vExp = (float *)malloc(sizeof(float)*(nCols+1)*nRows*nGPUs);	// store explicit V velocity

	float *Global_uCoeff = (float *)malloc(sizeof(float)*(nCols+1)*nRows*nGPUs);	// store U velocity coefficients
	float *Global_vCoeff = (float *)malloc(sizeof(float)*(nCols+1)*nRows*nGPUs);	// store V velocity coefficients

	#pragma omp parallel for schedule(auto)
	for(int myImg = 0; myImg<opts.nImg; myImg++){

		// Start datastructures

		simulationInfo simInfo;
		convInfo *Conv;

		Conv = (convInfo *)malloc(sizeof(convInfo)*opts.MaxIterGlobal);

		memset(Conv, 0, sizeof(convInfo)*opts.MaxIterGlobal);
		
		// Get thread index

		int threadIdx = omp_get_thread_num();

		printf("Thread idx = %d, Img num = %d\n", threadIdx, myImg);

		hipSetDevice(threadIdx);

		// Read 2D Input Image

		int width, height, channel;
		unsigned char* targetImage;
		char filename[100];
		char convFile[100];

		sprintf(filename, "%05d.jpg", myImg);

		readImage(&targetImage, &width, &height, &channel, filename);

		if (channel != 1){
			printf("Error: please enter a grascale image with 1 channel.\n Current number of channels = %d\n", channel);
		}

		simInfo.porosity = calcPorosity(targetImage, width, height);

		if(opts.verbose == 1){
			std::cout << "Image Parameters:" << std::endl;
			std::cout <<  "\n--------------------------------------" << std::endl;
			std::cout << "Width (pixels) = " << width << " Height (pixels) = " << height << " Channel = " << channel << std::endl;
			std::cout << "Porosity = " << simInfo.porosity << std::endl;
		}

		// Define mesh related parameters

		simInfo.numCellsX = width*opts.MeshAmp;			// Simulation Grid width in number of cells
		simInfo.numCellsY = height*opts.MeshAmp;		// Simulation Grid height in number of cells
		simInfo.nElements = simInfo.numCellsY*simInfo.numCellsX;	// Number of elements (total)
		simInfo.dx = opts.DomainWidth/simInfo.numCellsX;			// dx
		simInfo.dy = opts.DomainWidth/simInfo.numCellsY;			// dy

		unsigned int *Grid = Global_Grid + threadIdx * nElements;		// Array that will hold binary domain (solid vs fluid)

		// Mesh Amplify and decode image into binary matrix

		for(int i = 0; i<simInfo.numCellsY; i++){
			for (int j = 0; j<simInfo.numCellsX; j++){
				int targetIndex_Row = i/opts.MeshAmp;
				int targetIndex_Col = j/opts.MeshAmp;
				if(targetImage[targetIndex_Row*width + targetIndex_Col] < 150){
					Grid[i*simInfo.numCellsX + j] = 0; 			// black => fluid => 0 => void
				} else{
					Grid[i*simInfo.numCellsX + j] = 1;			// white => solid => 1 => material
				}
			}
		}

		// Free target_image since it is not needed anymore
		free(targetImage);

		// Flood Fill to eliminate non-participating media

		FloodFill(Grid, &simInfo);

		std::cout << "Flood Fill Successfull. Thread = " << threadIdx << std::endl;

		// Define arrays essential for the solution

		float *Pressure = Global_Pressure + threadIdx * nElements;
		float *U = Global_U + threadIdx * (nCols + 1)*nRows;
		float *V = Global_V + threadIdx * (nRows + 1)*nCols;

		float *uExp = Global_uExp + threadIdx * (nCols + 1)*nRows;
		float *vExp = Global_vExp + threadIdx * (nRows + 1)*nCols;

		float *uCoeff = Global_uCoeff + threadIdx * (nCols + 1)*nRows;
		float *vCoeff = Global_vCoeff + threadIdx * (nRows + 1)*nCols;

		std::cout << "Allocated arrays Successfull. Thread = " << threadIdx << std::endl;

		// Initialize arrays

		for(int row = 0; row<simInfo.numCellsY; row++){
			for(int col = 0; col< simInfo.numCellsX+1; col++){
				int index = row*(simInfo.numCellsX + 1) + col;
				if(col < simInfo.numCellsX){
					Pressure[row*(simInfo.numCellsX) + col] =  (1.0 - (float)col/(simInfo.numCellsX))*(opts.PL - opts.PR) + opts.PR;
				}
				U[index] = 0.01;
				V[index] = 0.0;
				uExp[index] = 0.01;
				vExp[index] = 0.0;
				uCoeff[index] = 0.0;
				vCoeff[index] = 0.0;
			}	
		}

		// start file to get convergence data

		if(convFlag == true)
		{
			sprintf(convFile, "ConvData_%05d.csv", myImg);
		}

		// Now we use the SUV-CUT algorithm to solve velocity-pressure coupled

		float RMS = 1.0;
		long int iter = 0;

		float PermTHR = 0.001;
		float PermOld = 1;
		float PermChange = 1;

		std::cout << "Start loop Successfull." << std::endl;

		while(iter < opts.MaxIterGlobal && RMS > opts.ConvergenceRMS && PermChange > PermTHR){

			/*
				SUV-CUT procedure:
				- Solve for explicit component of u and v velocities
				- Use explicit u and v to solve for pressure implicitly
				- Use pressure solutions to correct u and v explicitly

				- (optional) solve equations of state to update physical properties

				Repeat until converged.

			*/

			if(iter == 0){
				printf("Global Iter: %ld, Thread = %d\n\n", iter+1, threadIdx);
			}else if(iter % 10 == 0){
				printf("Global Iter: %ld, Thread = %d\n", iter+1, threadIdx);
				printf("Permeability: %f\n", simInfo.Perm);
				printf("Continuity RMS: %1.9f\n\n", RMS);
			}
			// std::cout << "Thread Num:" << omp_get_thread_num() << "Explicit Momentum Iter" << iter << std::endl;
			explicitMomentum(Grid, uExp, vExp, U, V, uCoeff, vCoeff, &opts, &simInfo);
			// std::cout << "Thread Num:" << omp_get_thread_num() << "implicitPressure Iter" << iter << std::endl;
			implicitPressure(Grid, uExp, vExp, uCoeff, vCoeff, Pressure, &opts, &simInfo);
			// std::cout << "Thread Num:" << omp_get_thread_num() << "Momentum Correction Iter" << iter << std::endl;
			momentumCorrection(Grid, uExp, vExp, U, V, uCoeff, vCoeff, Pressure, &opts, &simInfo);

			RMS = ResidualContinuity(U, V, &opts, &simInfo);

			PermCalc(U, &opts, &simInfo);

			// Update our convergence file

			Conv[iter].iter = iter;
			Conv[iter].Perm = simInfo.Perm;
			Conv[iter].Residual = RMS;
			Conv[iter].PermChange = PermChange;


			// Calculate Perm change to see if it flatlined over 100 iterations
			if(iter % 100 == 0){
				PermChange = fabs((simInfo.Perm - PermOld)/simInfo.Perm);
				PermOld = simInfo.Perm;
			}

			iter++;
		}

		// Print PUV Map

		if(opts.printMaps == 1){
			printPUVmaps(Pressure, U, V, &opts, &simInfo, myImg);
		}

		// Print batch output

		printBatchOut(&opts, &simInfo, myImg, iter, RMS);

		// Print convergence data if user wants it
		if(convFlag == true){
			FILE *CONV;
			CONV = fopen(convFile, "w+");
			fprintf(CONV, "iter,K,R,Kchange\n");
			for(int i=0; i<opts.MaxIterGlobal; i++){
				fprintf(CONV,"%ld,%f,%f,%f\n", Conv[i].iter, Conv[i].Perm, Conv[i].Residual, Conv[i].PermChange);
			}
			fclose(CONV);
		}

		free(Conv);

	}

	return 0;
}