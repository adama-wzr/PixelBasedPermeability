#include "Perm2D.cuh"

int main(void){

	// More efficient printing with parallel computing/Linux
	fflush(stdout);

	// Parse user entered options

	options opts;	// struct to hold options
	simulationInfo simInfo;		// Struct to hold constants, variables, and results intrinsic to the simulation

	char inputFilename[100];

	sprintf(inputFilename, "input.txt");

	readInputFile(inputFilename, &opts);

	// Read 2D Input Image

	int width, height, channel;
	unsigned char* targetImage;

	readImage(&targetImage, &width, &height, &channel, opts.inputFilename);

	simInfo.porosity = calcPorosity(targetImage, width, height);


	return 0;
}